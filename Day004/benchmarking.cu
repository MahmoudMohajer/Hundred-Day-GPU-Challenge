#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include "cuda_utils.h"

__global__ void vectorAdd(const float *a, const float *b,float *c, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x; 

    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() { 
    const int size = 500000000;  
    const int bytes = size * sizeof(float); // around 0.5GB for each array 
    float *a_h = (float*)malloc(bytes);
    float *b_h = (float*)malloc(bytes);
    float *c_h = (float*)malloc(bytes);

    for (int i=0; i < size; i++) {
        a_h[i] = i * 1.23; 
        b_h[i] = i * 2.12;
    }

    // allocating memory for host arrays 
    float *a_d, *b_d, *c_d; 
    cudaCheckError(hipMalloc((void**)&a_d, bytes));
    cudaCheckError(hipMalloc((void**)&b_d, bytes));
    cudaCheckError(hipMalloc((void**)&c_d, bytes));

    hipEvent_t start, stop, startKernel, stopKernel; 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&startKernel);
    hipEventCreate(&stopKernel);
    
    // copy to vram 
    hipEventRecord(start, 0);
    cudaCheckError(hipMemcpy(a_d, a_h, bytes, hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(b_d, b_h, bytes, hipMemcpyHostToDevice));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
    float memcpyHtoDTime = 0;
    hipEventElapsedTime(&memcpyHtoDTime, start, stop);
    printf("Host to Device memcpy time: %f ms\n", memcpyHtoDTime);

    int threadsPerBlock = 256;
    int BlocksPerGrid = (size + threadsPerBlock -1) / threadsPerBlock;

    // launching the kernel 
    hipEventRecord(startKernel, 0);
    vectorAdd<<<BlocksPerGrid, threadsPerBlock>>>(a_d, b_d, c_d, size);
    hipEventRecord(stopKernel, 0);
    hipEventSynchronize(stopKernel);
    float kernel_time = 0; 
    hipEventElapsedTime(&kernel_time, startKernel, stopKernel);
    printf("Kernel operation time: %f ms\n", kernel_time);
    cudaCheckError(hipGetLastError());

    cudaCheckError(hipDeviceSynchronize());
    
    hipEventRecord(start, 0);
    cudaCheckError(hipMemcpy(c_h, c_d, bytes, hipMemcpyDeviceToHost));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float memcpyDtoHTime = 0;
    hipEventElapsedTime(&memcpyDtoHTime, start, stop); 
    printf("Elapsed time copying from Device to Host: %f ms\n", memcpyDtoHTime);

    
    
    printf("The addition was done successfully\n");

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(startKernel);
    hipEventDestroy(stopKernel);
    cudaCheckError(hipFree(a_d));
    cudaCheckError(hipFree(b_d));
    cudaCheckError(hipFree(c_d));

    free(a_h);
    free(b_h);
    free(c_h);


}