
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__
void brightness_adjust(const unsigned char *input, unsigned char *output, int width, int height, int brightness_offset){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < height && col < width){
        int idx = row * width + col;
        int pixel = input[idx] + brightness_offset;
        if (pixel > 255) pixel = 255;
        if (pixel < 0) pixel = 0;
        output[idx] = (unsigned char) pixel;
    }
}

int main(int argc, char** argv) {
    if (argc < 4) {
        printf("Usage: %s input.pgm output.pgm\n", argv[0]);
        return 1;
    }

    FILE *fp = fopen(argv[1], "rb");
    if (!fp) {
        perror("Error opening input file\n");
        return 1;
    }

    char format[3];
    if (fscanf(fp, "%2s", format) != 1) {
        fprintf(stderr, "Error reading PGM format\n");
        fclose(fp);
        return 1;
    }
    if (format[0] != 'P' || format[1] != '5') {
        fprintf(stderr, "Only binary PGM (P5) is supported\n");
        fclose(fp);
        return 1;
    }

    int width, height, maxval;
    fscanf(fp, "%d %d",&width, &height);
    fscanf(fp, "%d", &maxval);

    if (maxval != 255) {
        fprintf(stderr, "Only maxval of 255 is supported\n");
        fclose(fp);
        return 1;
    }

    int size = width * height;
    unsigned char *h_input = (unsigned char *)malloc(size);
    if (!h_input){
        fprintf(stderr, "couldn't allocate memory for host input\n");
        fclose(fp);
        return 1;
    } 

    if (fread(h_input, 1, size, fp) != size) {
        fprintf(stderr, "The elements are less than than expected size\n");
        fclose(fp);
        return 1;
    }
    fclose(fp);

    unsigned char *h_output = (unsigned char *)malloc(size);
    if (!h_output){
        fprintf(stderr, "couldn't allocate memory for host output\n");
        free(h_input);
        return 1;
    }

    unsigned char *d_input, *d_output; 
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice); 

    int BLOCK_SIZE = 16;
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y); 

    int brightness_offset = atoi(argv[3]);
    brightness_adjust<<<grid, block>>>(d_input, d_output, width, height, brightness_offset); 

    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost); 

    fp = fopen(argv[2], "wb");
    if (!fp) {
        perror("Error opening output file");
        free(h_input);
        free(h_output);
        hipFree(d_input);
        hipFree(d_output);
        return 1;
    } 

    fprintf(fp, "P5\n%d %d\n255\n", width, height);
    fwrite(h_output, 1, size, fp);
    fclose(fp);

    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}