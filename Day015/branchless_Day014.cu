
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_CHECK(call)\
    do \
    { \
        hipError_t error = call; \
        if (error != hipSuccess) \
        { \
            fprintf(stderr, "Error at %s:%d %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
} while (0);

__global__ 
void sumArray(float *input, float *result, int n) {
    extern __shared__ float sdata[]; 
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 

    sdata[tid] = input[idx] * (idx < n); 

    __syncthreads(); 
    for (int stride = blockDim.x/2; stride > 0; stride /= 2) {
        int offset = (tid < stride) * stride; 
        sdata[tid] +=  sdata[tid + offset] * (offset != 0);

        __syncthreads();
    }

    if (tid == 0) {
        result[blockIdx.x] = sdata[0];
    }
}

int main() {
    int n = 1 << 28; 
    int bytes = n * sizeof(float); 

    float *h_input = (float*)malloc(bytes); 
    float h_result;

    for (int i=0; i < n; i++) h_input[i] = 1.0f;

    float *d_input, *d_result; 
    CUDA_CHECK(hipMalloc((void**)&d_input, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_result, bytes));

    CUDA_CHECK(hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice));

    int block_size = 256; 
    int current_size = n; 
    int shared_memory = block_size * sizeof(float);
    float *input = d_input;
    float *result = d_result;
    
    while (current_size != 1) {
        int num_blocks = (current_size + block_size - 1) / block_size;
        dim3 grid(num_blocks);
        sumArray<<<grid, block_size, shared_memory>>>(input, result, current_size);
        CUDA_CHECK(hipGetLastError());

        float *temp = input;
        input = result; 
        result = temp; 

        current_size = num_blocks;
    }

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(&h_result, input, sizeof(float), hipMemcpyDeviceToHost));

    printf("the Sum is %f \n", h_result);

    return 0;

}