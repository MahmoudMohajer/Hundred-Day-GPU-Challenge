#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Vector addition kernel with increased computation
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float sum = a[idx] + b[idx];
        for (int i = 0; i < 1000; i++) {  // Heavy computation
            sum += sinf(sum) * cosf(sum);
        }
        c[idx] = sum;
    }
}

int main() {
    const int N = 1 << 28;  // 268,435,456 elements
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float milliseconds = 0;

    
    
    // Host arrays (pinned memory)
    float *h_a, *h_b, *h_c;
    CUDA_CHECK(hipHostAlloc((void**)&h_a, N * sizeof(float), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc((void**)&h_b, N * sizeof(float), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc((void**)&h_c, N * sizeof(float), hipHostMallocDefault));

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }
    
    // Record start time for total execution
    CUDA_CHECK(hipEventRecord(start));
    
    // Device arrays
    float *d_a, *d_b, *d_c;
    CUDA_CHECK(hipMalloc(&d_a, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_b, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_c, N * sizeof(float)));

    // Synchronous memory transfers and kernel launch
    CUDA_CHECK(hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice));

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    
    // Synchronize device and copy back results
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost));

    // Record stop time and calculate total execution time
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Total execution time: %.2f milliseconds\n", milliseconds);

    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
    CUDA_CHECK(hipHostFree(h_a));
    CUDA_CHECK(hipHostFree(h_b));
    CUDA_CHECK(hipHostFree(h_c));

    // Cleanup events
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
}