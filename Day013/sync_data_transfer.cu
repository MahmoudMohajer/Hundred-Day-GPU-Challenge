#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Vector addition kernel with increased computation
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float sum = a[idx] + b[idx];
        for (int i = 0; i < 1000; i++) {  // Heavy computation
            sum += sinf(sum) * cosf(sum);
        }
        c[idx] = sum;
    }
}

int main() {
    const int N = 1 << 28;  // 268,435,456 elements
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Host arrays (pinned memory)
    float *h_a, *h_b, *h_c;
    CUDA_CHECK(hipHostAlloc((void**)&h_a, N * sizeof(float), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc((void**)&h_b, N * sizeof(float), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc((void**)&h_c, N * sizeof(float), hipHostMallocDefault));

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }

    // Device arrays
    float *d_a, *d_b, *d_c;
    CUDA_CHECK(hipMalloc(&d_a, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_b, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_c, N * sizeof(float)));

    // Timing variables
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Record start time
    CUDA_CHECK(hipEventRecord(start));

    // Synchronous memory transfers and kernel launch
    CUDA_CHECK(hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice));
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    CUDA_CHECK(hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost));

    // Record stop time
    CUDA_CHECK(hipEventRecord(stop));

    // Synchronize device
    CUDA_CHECK(hipDeviceSynchronize());

    // Calculate and print execution time
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Sync Execution time: %.2f ms\n", milliseconds);

    // Cleanup
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
    CUDA_CHECK(hipHostFree(h_a));
    CUDA_CHECK(hipHostFree(h_b));
    CUDA_CHECK(hipHostFree(h_c));

    return 0;
}