#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Vector addition kernel with increased computation
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float sum = a[idx] + b[idx];
        for (int i = 0; i < 1000; i++) {  // Heavy computation
            sum += sinf(sum) * cosf(sum);
        }
        c[idx] = sum;
    }
}

int main() {
    const int N = 1 << 28;  // 268,435,456 elements
    const int STREAMS = 4;  // Number of streams
    const int SEGMENT_SIZE = N / STREAMS;
    const int BYTES = SEGMENT_SIZE * sizeof(float);

    // Host arrays (pinned memory)
    float *h_a, *h_b, *h_c;
    CUDA_CHECK(hipHostAlloc((void**)&h_a, N * sizeof(float), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc((void**)&h_b, N * sizeof(float), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc((void**)&h_c, N * sizeof(float), hipHostMallocDefault));

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }

    // Device arrays
    float *d_a, *d_b, *d_c;
    CUDA_CHECK(hipMalloc(&d_a, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_b, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_c, N * sizeof(float)));

    // Create streams
    hipStream_t streams[STREAMS];
    for (int i = 0; i < STREAMS; i++) {
        CUDA_CHECK(hipStreamCreate(&streams[i]));
    }

    // Timing variables
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Record start time
    CUDA_CHECK(hipEventRecord(start));

    // Launch kernels and transfers using streams
    int threadsPerBlock = 256;
    int blocksPerGrid = (SEGMENT_SIZE + threadsPerBlock - 1) / threadsPerBlock;
    for (int i = 0; i < STREAMS; i++) {
        int offset = i * SEGMENT_SIZE;
        
        // Asynchronous memory transfers
        CUDA_CHECK(hipMemcpyAsync(&d_a[offset], &h_a[offset], BYTES,
                                 hipMemcpyHostToDevice, streams[i]));
        CUDA_CHECK(hipMemcpyAsync(&d_b[offset], &h_b[offset], BYTES,
                                 hipMemcpyHostToDevice, streams[i]));
        
        // Kernel launch
        vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>
            (&d_a[offset], &d_b[offset], &d_c[offset], SEGMENT_SIZE);
        
        // Asynchronous memory transfer back to host
        CUDA_CHECK(hipMemcpyAsync(&h_c[offset], &d_c[offset], BYTES,
                                 hipMemcpyDeviceToHost, streams[i]));
    }

    // Record stop time
    CUDA_CHECK(hipEventRecord(stop));

    // Synchronize all streams
    for (int i = 0; i < STREAMS; i++) {
        CUDA_CHECK(hipStreamSynchronize(streams[i]));
    }

    // Calculate and print execution time
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Async Execution time: %.2f ms\n", milliseconds);

    // Cleanup
    for (int i = 0; i < STREAMS; i++) {
        CUDA_CHECK(hipStreamDestroy(streams[i]));
    }
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
    CUDA_CHECK(hipHostFree(h_a));
    CUDA_CHECK(hipHostFree(h_b));
    CUDA_CHECK(hipHostFree(h_c));

    return 0;
}